#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "cuda_helper_funcs.h"
#include "RGB.h"

#include <iostream>
using namespace std;

// Block dimensions
#define X_BLOCK_SIZE	32
#define Y_BLOCK_SIZE	32

// Shared variables: fast memory for each block
__shared__ float red_share[X_BLOCK_SIZE][Y_BLOCK_SIZE], green_share[X_BLOCK_SIZE][Y_BLOCK_SIZE], blue_share[X_BLOCK_SIZE][Y_BLOCK_SIZE];

__global__ void accumulate_colors(RGB *d_pixels, int height, int width, float3 *total_colors)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x; // width index
	int y = blockIdx.y * blockDim.y + threadIdx.y; // height index

	if (y >= height || x >= width) // index is not withing image
		return;

	int index = y * width + x; // index into pixel buffer

	int totalThreads = blockDim.x;
	int me = threadIdx.x;

	red_share[me][threadIdx.y] = d_pixels[index].red;
	green_share[me][threadIdx.y] = d_pixels[index].green;
	blue_share[me][threadIdx.y] = d_pixels[index].blue;

	__syncthreads(); //language extension not actually a function

	while (totalThreads > 1){
		int buddy_line = totalThreads / 2;
		if (me < buddy_line){
			int buddy = me + buddy_line;
			red_share[me][threadIdx.y] += red_share[buddy][threadIdx.y];
			green_share[me][threadIdx.y] += green_share[buddy][threadIdx.y];
			blue_share[me][threadIdx.y] += blue_share[buddy][threadIdx.y];
		}
		__syncthreads();
		totalThreads /= 2;
	}

	if (me == 0){
		int myblock = blockIdx.x + blockIdx.y * gridDim.x;
		atomicAdd(&total_colors[myblock].x, red_share[me][threadIdx.y]);
		atomicAdd(&total_colors[myblock].y, green_share[me][threadIdx.y]);
		atomicAdd(&total_colors[myblock].z, blue_share[me][threadIdx.y]);
	}
}

__host__ void d_compute_component_average(RGB *pixel, int height, int width)
{
	RGB *d_pixel; // Part 1: pointer to array for storing image pixel on device

	hipMalloc(&d_pixel, height * width * sizeof(RGB)); // Part 2: allocate space on device
	hipMemcpy(d_pixel, pixel, height * width * sizeof(RGB), hipMemcpyHostToDevice); // Part 3: copy image to device

	dim3 grid, block;
	block.x = X_BLOCK_SIZE;
	block.y = Y_BLOCK_SIZE;
	grid.x = calcBlockDim(width, block.x);
	grid.y = calcBlockDim(height, block.y);

	float3 *d_total_colors; // part 1: pointer to array for storing partial sums from each block
							// float3::x red
							// float3::y green
							// float3::z blue
	hipMalloc(&d_total_colors, grid.x * grid.y * sizeof(float3)); // part 2: allocate space on device
	hipMemset(d_total_colors, 0, grid.x * grid.y * sizeof(float3)); // part 3: initialize array to zero

	accumulate_colors << <grid, block >> >(d_pixel, height, width, d_total_colors);

	float3 *h_total_colors = new float3[grid.x * grid.y];	// allocate memory of host to collect partial sums from device
	hipMemcpy(h_total_colors, d_total_colors, grid.x * grid.y * sizeof(float3), hipMemcpyDeviceToHost); // copy partial sums from device

	double total_green = 0, total_red = 0, total_blue = 0;
	for (int i = 0; i < grid.x*grid.y; ++i) { // sum up all the partial sums returned from each block
		total_red += h_total_colors[i].x;
		total_green += h_total_colors[i].y;
		total_blue += h_total_colors[i].z;
	}

	// Display primary color averages
	cout << "Red average: " << total_red / (height*width) << endl;
	cout << "Green average: " << total_green / (height*width) << endl;
	cout << "Blue average: " << total_blue / (height*width) << endl;
}