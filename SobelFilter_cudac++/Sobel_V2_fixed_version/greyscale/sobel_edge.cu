#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "cuda_helper_funcs.h"
#include "RGB.h"
#include <math.h>
#include <vector>

/**
* Helper function to calculate the greyscale value based on R, G, and B
*/
__device__ int greyscale(BYTE red, BYTE green, BYTE blue)
{
	int grey = 0.3 * red + 0.59 * green + 0 * 11 * blue; // calculate grey scale
	return min(grey, 255);
}

/**
* Kernel for executing on GPY
*/
__global__ void greyscaleKernel(RGB *d_pixels, int height, int width)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x; // width index
	int y = blockIdx.y * blockDim.y + threadIdx.y; // height index

	if (y >= height || y >= width) // thread is not within image
		return;

	int index = y * width + x;

	int grey = greyscale(d_pixels[index].red, d_pixels[index].green, d_pixels[index].blue); // calculate grey scale

	d_pixels[index].red = grey;
	d_pixels[index].green = grey;
	d_pixels[index].blue = grey;
}

/**
*	Host function for launching greyscale kernel
*/
__host__ void d_convert_greyscale(RGB *pixel, int height, int width)
{
	RGB *d_pixel;

	hipMalloc(&d_pixel, height * width * sizeof(RGB)); // Allocate memory on device to store pixels
	hipMemcpy(d_pixel, pixel, height * width * sizeof(RGB), hipMemcpyHostToDevice); // Copy pixels to device

	dim3 grid, block;
	block.x = 16;
	block.y = 16;
	grid.x = calcBlockDim(width, block.x);
	grid.y = calcBlockDim(height, block.y);

	greyscaleKernel << <grid, block >> >(d_pixel, height, width);

	hipMemcpy(pixel, d_pixel, height * width * sizeof(RGB), hipMemcpyDeviceToHost); // Copy results back to the host
}

#define P(x,y)	d_pixels[(y)* width + (x)].blue
/**
* Helper function for computing the sobel value at pixel (x,y)
*/
__device__ int sobel_filter(int x, int y, RGB *d_pixels, int height, int width)
{
	//convolution 
	/*int Gx = -(d_pixels[(y - 1) * width + (x - 1)].blue) - 2 * (d_pixels[(y)* width + (x - 1)].blue) - (d_pixels[(y+1)* width + (x-1)].blue) 
		+ (d_pixels[(y + 1)* width + (x - 1)].blue) * (d_pixels[(y)* width + (x + 1)].blue) + (d_pixels[(y+1)* width + (x+1)].blue);

	int Gy = (d_pixels[(y - 1)* width + (x - 1)].blue) + 2 * (d_pixels[(y - 1)* width + (x)].blue) + (d_pixels[(y-1)* width + (x+1)].blue)
		- (d_pixels[(y + 1)* width + (x - 1)].blue) - 2 * (d_pixels[(y + 1)* width + (x)].blue) - (d_pixels[(y+1)* width + (x+1)].blue);
*/
	//Eds c++ macro trick much better

	int Gx = -P(x-1,y-1) - 2 * P(x-1,y) - P(x-1,y+1)
		+ P(x-1,y+1) + 2 * P(x+1,y) + P(x+1,y+1);
	
	int Gy = P(x - 1, y - 1) + 2 * P(x, y - 1) + P(x + 1, y - 1)
		- P(x - 1, y + 1) - 2 * P(x, y + 1) - P(x + 1, y + 1);
		
	int sum = abs(Gx) + abs(Gy);

	return min(sum,255);
}

/**
* Kernel for computing sobel filter 
*/
__global__ void sobelKernel(RGB *d_pixels, int height, int width)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x; // width index
	int y = blockIdx.y * blockDim.y + threadIdx.y; // height index

	if (y >= height || x >= width) // thread is not within image
		return;

	if (x > 0 && y > 0 && y < height - 1 && x < width - 1) { // valid sobel range
		d_pixels[(y)* width + (x)].green = sobel_filter(x, y, d_pixels, height, width); //store sobel values in the pixel's green variable
	}
}

/**
* SECOND!!! Kernel for updating red and blue components with the sobel filter values stored in green
*/
__global__ void componentUpdateKernel(RGB *d_pixels, int height, int width)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x; // width index
	int y = blockIdx.y * blockDim.y + threadIdx.y; // height index

	if (y >= height || x >= width) // thread is not within image
		return;

	if (x > 0 && y > 0 && y < height - 1 && x < width - 1) { // See sobelKernel
		d_pixels[(y)* width + (x)].red = d_pixels[(y)* width + (x)].green; // assign sobel value to red and blue variables 
		d_pixels[(y)* width + (x)].blue = d_pixels[(y)* width + (x)].green;
	}
}

/**
*	Host function for launching the sobel kernel
*/
__host__ void d_sobel_edge(RGB *pixel, int height, int width)
{
	RGB *d_pixel;

	hipMalloc(&d_pixel, height * width * sizeof(RGB)); 
	hipMemcpy(d_pixel, pixel, height * width * sizeof(RGB), hipMemcpyHostToDevice); 

	dim3 grid, block;
	block.x = 16;
	block.y = 16;
	grid.x = calcBlockDim(width, block.x);
	grid.y = calcBlockDim(height, block.y);

	greyscaleKernel << <grid, block >> >(d_pixel, height, width); 
	sobelKernel << <grid, block >> >(d_pixel, height, width); 
	//second sobel kernel comes into play 
	componentUpdateKernel << <grid, block >> >(d_pixel, height, width);

	hipMemcpy(pixel, d_pixel, height * width * sizeof(RGB), hipMemcpyDeviceToHost); 

	hipDeviceSynchronize();
}