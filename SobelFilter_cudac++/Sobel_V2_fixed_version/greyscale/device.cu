
#include <hip/hip_runtime.h>
//#include "cuda_runtime.h"
//#include "device_launch_parameters.h"
//#include "cuda_helper_funcs.h"
//#include "RGB.h"
//
///**
//* Helper function to calculate the greyscale value based on R, G, and B
//*/
//__device__ int greyscale(BYTE red, BYTE green, BYTE blue)
//{
//	int grey = 0.3 * red + 0.59 * green + 0 * 11 * blue; // calculate grey scale
//	return min(grey, 255);
//}
//
///**
//* Kernel for executing on GPY
//*/
//__global__ void greyscaleKernel(RGB *d_pixels, int height, int width)
//{
//	int x = blockIdx.x * blockDim.x + threadIdx.x; // width index
//	int y = blockIdx.y * blockDim.y + threadIdx.y; // height index
//
//	if (y >= height || y >= width) // thread is not within image
//		return;
//
//	int index = y * width + x;
//
//	int grey = greyscale(d_pixels[index].red, d_pixels[index].green, d_pixels[index].blue); // calculate grey scale
//
//	d_pixels[index].red = grey;
//	d_pixels[index].green = grey;
//	d_pixels[index].blue = grey;
//}
//
///**
//*	Host function for launching greyscale kernel
//*/
//__host__ void d_convert_greyscale(RGB *pixel, int height, int width)
//{
//	RGB *d_pixel;
//
//	cudaMalloc(&d_pixel, height * width * sizeof(RGB));
//	cudaMemcpy(d_pixel, pixel, height * width * sizeof(RGB), cudaMemcpyHostToDevice);
//
//	dim3 grid, block;
//	block.x = 16;
//	block.y = 16;
//	grid.x = calcBlockDim(width, block.x);
//	grid.y = calcBlockDim(height, block.y);
//
//	greyscaleKernel << <grid, block >> >(d_pixel, height, width);
//
//	cudaMemcpy(pixel, d_pixel, height * width * sizeof(RGB), cudaMemcpyDeviceToHost);
//}